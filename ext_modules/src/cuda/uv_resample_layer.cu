#include <torch/extension.h>

#include "cuda/resample.cuh"

namespace spherical {

namespace cuda {

torch::Tensor EquirectangularToTangentImages(
    torch::Tensor input, torch::Tensor quad_idx, torch::Tensor tex_uv,
    int numTextures, int textureHeight, int textureWidth,
    InterpolationType interpolation) {
  // Useful dimensions to have
  const int64_t batchSize   = input.size(0);
  const int64_t channels    = input.size(1);
  const int64_t inputHeight = input.size(2);
  const int64_t inputWidth  = input.size(3);

  // Initialize output and index mask
  torch::Tensor output = torch::zeros(
      {batchSize, channels, numTextures, textureHeight, textureWidth},
      input.options());

  // Call the CUDA kernel once per batch
  for (int b = 0; b < batchSize; b++) {
    ResampleToUVMapsLauncher(input[b], quad_idx, tex_uv, channels, numTextures,
                             textureHeight, textureWidth, inputHeight,
                             inputWidth, interpolation, output[b]);
  }

  return output;
}  // namespace cuda

torch::Tensor EquirectangularFromTangentImages(
    torch::Tensor grad_output, torch::Tensor quad_idx, torch::Tensor tex_uv,
    InterpolationType interpolation) {
  // Useful dimensions to have
  const int64_t batchSize     = grad_output.size(0);
  const int64_t channels      = grad_output.size(1);
  const int64_t numTextures   = grad_output.size(2);
  const int64_t textureHeight = grad_output.size(3);
  const int64_t textureWidth  = grad_output.size(4);
  const int64_t inputHeight   = quad_idx.size(0);
  const int64_t inputWidth    = quad_idx.size(1);

  // Initialize output and index mask
  torch::Tensor input = torch::zeros(
      {batchSize, channels, inputHeight, inputWidth}, grad_output.options());

  // Call the CUDA kernel once per batch
  for (int b = 0; b < batchSize; b++) {
    ResampleFromUVMapsLauncher(
        grad_output[b], quad_idx, tex_uv, channels, numTextures, textureHeight,
        textureWidth, inputHeight, inputWidth, interpolation, input[b]);
  }

  return input;
}

}  // namespace cuda

}  // namespace spherical